#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <ctime>

using namespace std;

//__global__ : ����̽����� ����Ǵ� �Լ� �ۼ�
__global__ void AddInts(int *a, int *b, int count) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < count) {
		a[id] += b[id];
	}
}


main() {
	srand(time(NULL));

	//count ��Ʈ�� ����.
	int count = 10000; 
	
	//host pointer. int �迭 ����.
	int *h_a = new int[count];
	int *h_b = new int[count];

	for (int i = 0; i < count; i++) {
		h_a[i] = rand() % 1000;
		h_b[i] = rand() % 1000;
	}

	//AddInts ���� �� ������
	cout << "Prior to addition : " << endl;
	for (int i = 0; i < 5; i++) {
		cout << h_a[i] << " " << h_b[i] << endl;
	}


	//device pointer.
	int *d_a, *d_b;

	//Malloc �Ҵ� @ device pointer
	if (hipMalloc(&d_a, sizeof(int) * count) != hipSuccess) {
		cout << "Nope!";
		return 0;
	}

	if (hipMalloc(&d_b, sizeof(int) * count) != hipSuccess) {
		cout << "Nope!";
		hipFree(d_a);
		return 0;
	}

	//data copy : host to device
	if (hipMemcpy(d_a, h_a, sizeof(int) * count, hipMemcpyHostToDevice) != hipSuccess) {
		cout << "cout not copy!" << endl;
		hipFree(d_a);
		hipFree(d_b);
		return 0;
	}

	if (hipMemcpy(d_b, h_b, sizeof(int) * count, hipMemcpyHostToDevice) != hipSuccess) {
		cout << "cout not copy!" << endl;
		hipFree(d_a);
		hipFree(d_b);
		return 0;
	}

	//AddInts ����
	AddInts <<<count / 256 + 1, 256>>> (d_a, d_b, count);


	//���н� �޸� ��ȯ �� ����
	if (hipMemcpy(h_a, d_a, sizeof(int) * count, hipMemcpyDeviceToHost) != hipSuccess) {
		delete[] h_a;
		delete[] h_b;
		hipFree(d_a);
		hipFree(d_b);
		cout << "Nope!" << endl;
		return 0;
	}

	//������ ��� ���
	for (int i = 0; i < 5; i++) {
		cout << "It's" << h_a[i] << endl;
	}


	//����̽��� �޸� ��ȯ
	hipFree(d_a);
	hipFree(d_b);

	delete[] h_a;
	delete[] h_b;

	system("pause");

	return 0; 
}
