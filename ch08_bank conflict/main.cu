#include "hip/hip_runtime.h"
#include <iostream>
#include <conio.h>
#include <hip/hip_runtime.h>

using namespace std;

struct Pehw{
    float x, y, z, w;
};

/*
__global__ void MyKernel(unsigned long long* time)
{
    __shared__ float shared[1024];
    unsigned long long startTime = clock();

    //code
    //shared[0]++;
    //shared[threadIdx.x]++;
    //shared[threadIdx.x * 2]++;
    shared[threadIdx.x * 32]++;

    unsigned long long finishTime = clock();
    *time = (finishTime - startTime);
}
*/

__global__ void MyKernel(unsigned long long* time)
{
    __shared__ Pehw shared[1024];
    unsigned long long startTime = clock();

    //code
    //shared[0]++;
    //shared[threadIdx.x]++;
    //shared[threadIdx.x * 2]++;
    
    shared[threadIdx.x].x++;

    unsigned long long finishTime = clock();
    *time = (finishTime - startTime);
}

int main(){
    unsigned long long time;
    unsigned long long* d_time;
    hipMalloc(&d_time, sizeof(unsigned long long));

    for(int i = 0; i < 10; i++){
        MyKernel<<<1, 32>>>(d_time);
        hipMemcpy(&time, d_time, sizeof(unsigned long long), hipMemcpyDeviceToHost);

        cout << "time : " << (time-14)/32 << endl;

        cout << endl;
    }

    hipFree(d_time);

    _getch();
    hipDeviceReset(); 
    return 0;
}