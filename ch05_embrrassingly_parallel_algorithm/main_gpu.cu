#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <ctime>
#include <>

#include "FindClosestGPU.h"

using namespace std;

int main(){
    
	////////////////////////////////////////////////////////////////////////////////////////////////
	////////////////////////////////////////////////////////////////////////////////////////////////

	cout << "MAKING RANDOM POINT ARRAY.." << endl;
	const int count = 10000;

	int *indexOfClosest = new int[count];
    float3 *points = new float3[count];
    
    int *d_indexOfClosest;
    float3 *d_points;


	for (int i = 0; i < count; i++) {
		points[i].x = (float)((rand() % 10000) - 5000);
		points[i].y = (float)((rand() % 10000) - 5000);
		points[i].z = (float)((rand() % 10000) - 5000);
    }
    
    hipMalloc(&d_points, sizeof(float3)*count);
    hipMemcpy(d_points, points, sizeof(float3)*count, hipMemcpyHostToDevice);
    hipMalloc(&d_indexOfClosest, sizeof(int)*count);
    
	////////////////////////////////////////////////////////////////////////////////////////////////
	////////////////////////////////////////////////////////////////////////////////////////////////

	cout << endl << "GPU PROCESSING.." << endl;

    long fastest = 1000000;
    
    for(int q = 0; q < 20; q++){
        long startTime = clock();

        FindClosestGPU<<<(count/32)+1, 32>>>(d_points, d_indexOfClosest, count);
        hipMemcpy(indexOfClosest, d_indexOfClosest, sizeof(int)*count, hipMemcpyDeviceToHost);

        long finishTime = clock();
        

        cout << "Run : " << q << " took " << (finishTime - startTime) << " millis" << endl;
        if((finishTime - startTime) < fastest) fastest = (finishTime - startTime);
    }
    for (int i = 0; i < 10; i++) cout << i << "." << indexOfClosest[i] << endl;

    delete[] indexOfClosest;
    delete[] points;
    hipFree(d_points);
    hipFree(d_indexOfClosest);
    hipDeviceReset();

    return 0;
}