#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <ctime>
#include <>

#include "FindClosestCPU.h"

using namespace std;

int main() {

	////////////////////////////////////////////////////////////////////////////////////////////////
	////////////////////////////////////////////////////////////////////////////////////////////////

	cout << "MAKING RANDOM POINT ARRAY.." << endl;
	const int count = 10000;

	int *indexOfClosest = new int[count];
	float3 *points = new float3[count];
	
	for (int i = 0; i < count; i++) {
		points[i].x = (float)((rand() % 10000) - 5000);
		points[i].y = (float)((rand() % 10000) - 5000);
		points[i].z = (float)((rand() % 10000) - 5000);
	}

	////////////////////////////////////////////////////////////////////////////////////////////////
	////////////////////////////////////////////////////////////////////////////////////////////////

	cout << endl << "CPU PROCESSING.." << endl;

	long fastest = 1000000;

	for (int q = 0; q < 20; q++) {
		long startTime = clock();

		FindClosestCPU(points, indexOfClosest, count);

		long finishTime = clock();

		cout << "Run : " << q << " took " << (finishTime - startTime) << " millis" << endl;
		if ((finishTime - startTime) < fastest) fastest = (finishTime - startTime);
	}

	cout << "Fastest time : " << fastest << endl;

	cout << "Final result : " << endl;
	for (int i = 0; i < 10; i++) cout << i << "." << indexOfClosest[i] << endl;


	////////////////////////////////////////////////////////////////////////////////////////////////
	////////////////////////////////////////////////////////////////////////////////////////////////


	delete[] indexOfClosest;
	delete[] points;

	_getch();

	return 0;

}